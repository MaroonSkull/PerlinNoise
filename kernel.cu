#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <iostream>
#include <stdint.h>

// Объявляем функции
template<typename T>
hipError_t Perlin1DWithCuda(T *res, const T *k, T step, int numSteps, int controlPoints, int resultDotsCols, int octaveNum);
void framebuffer_size_callback(GLFWwindow *window, int width, int height);
void processInput(GLFWwindow *window);

// Source of OpenGL vertex shader
const char *vertexShaderSource =	"#version 330 core\n"
									"layout (location = 0) in vec3 aPos;\n"
									"void main() {\n"
									"   gl_Position = vec4(aPos.x, aPos.y, aPos.z, 1.0);\n"
									"}\0";

// Source of fragment shader
const char *fragmentShaderSource =	"#version 330 core\n"
									"out vec4 FragColor;\n"
									"void main() {\n"
									"	FragColor = vec4(1.0f, 0.25f, 0.25f, 1.0f)\n;"
									"}\0";

/**
* линейная интерполяция точки t на промежутке [0, 1] между двумя прямыми с наклонами k0 и k1 соответственно.
* 
* \param k0 – значение наклона прямой в точке 0.
* \param k1 – значение наклона прямой в точке 1.
* \param t – точка, значение в которой интерполируется.
* 
* \return Результат интерполяции.
*/
template <typename T>
__device__ inline
T lerp_kernel(T k0, T k1, T t) {
	return fma(t, k1 - k0, k0); // (1-t)*k0 + t*k1 = k0 - t*k0 + t*k1 = t*(k1 - k0) + k0
}


/**
* Сигмоидальная функция из семейства smoothstep, используется для создания более интенсивного градиента шума. 
* Подробнее см. https://en.wikipedia.org/wiki/Smoothstep#Variations
* 
* \param x – значение градиента (он же t)
* 
* \return возвращает классический smootherstep(x). Используется оригинальный второй полином Кена Перлина.
*/
template <typename T>
__device__ inline
T smootherstep_kernel(T x) {
	return fma(static_cast<T>(6), x * x, fma(static_cast<T>(-15), x, static_cast<T>(10))) * x * x * x; // 6x^5 - 15x^4 + 10x^3 = x^3(6x^2 - 15x + 10)
}

/**
* Одна октава одномерного шума Перлина на промежутке [n, n+1] в точке t
* 
* \param res – массив с результатом вычисления шума перлина на оси.
* \param k – массив со значениями наклона уравнений в контрольных узлах.
* \param step – величина шага между точками, в которых вычисляется шум.
* \param numSteps – количество точек между контрольными узлами.
* 
* \return res – функция изменяет переданный массив (хранится в памяти GPU).
*/
template <typename T>
__global__
void Perlin1D_kernel(T *res, T *octave, const T *k, int size, T step, int numSteps, int octaveNum) {
	uint32_t id = blockIdx.x*blockDim.x+threadIdx.x;// [0..] – всего точек для просчёта
	uint32_t n = static_cast<T>(id) * step;			// 0 0 / 1 1 / 2 2 / .. – какие точки к каким контрольным точкам принадлежат
	uint32_t dotNum = id % numSteps;				// 0 1 / 0 1 / 0 1 / .. – какую позицию занимает точка между левой и правой функцией
	T t = dotNum * step;							// 0.33 0.66 / 0.33 0.66 / .. – численное значение точки для интерполяции
	t = smootherstep_kernel<T>(t);		// Применяем сигмоидальную функцию для сглаживания входного параметра и, следовательно, итогового шума
	T y0 = k[n] * t;					// kx+b (b = 0)
	T y1 = k[n+1] * (t - 1);			// kx+b (b = -k) = k(x-1)
	res[id] = lerp_kernel<T>(y0, y1, t);// Интерполяцией находим шум, пишем сразу в выходной массив
	
	// Если нужно вычислять октавы, делаем это
	if(octaveNum != 0) {
		// Сохраняем в глобальной памяти первую октаву шума
		if(id % 2 == 0)
			octave[id >> 1] = res[id] * 0.5;

		// синхронизируем выполнение, убеждаясь, что все данные октавы заполнены
		__syncthreads();

		// Применяем наложение октав, каждый раз основываясь на предыдущей октаве
		for(int j = 1; j <= octaveNum; j++) {
			int octavePov = 1 << j;
			for(int i = 0; i < octavePov; i++) {
				if((id < (i + 1) * size / octavePov) && (id >= i * size / octavePov))
					res[id] += octave[(id - i * size / octavePov) * (octavePov >> 1)] / (octavePov >> 1);
			}
		}
	}
}

int main() {
	constexpr int32_t controlPoints = 6;
	constexpr int32_t numSteps = 204; // пока что код хоста, запускающий cuda, не позволяет запускать более 1024 потоков
	constexpr int32_t octaveNum = 1;
	constexpr int32_t resultDotsCols = (controlPoints - 1) * numSteps; // 204 * (6-1) = 1020 потоков
	constexpr float step = 1.0f / numSteps;
	constexpr float k[controlPoints] = {.6f, -.3f, 1.0f, -.6f, -0.1, .6f}; // значения наклонов на углах отрезков (последний наклон равен первому)
	// Perlin noise coords
	float noise[resultDotsCols] = {};
	float vertices[3 * resultDotsCols] = {}; //x, y, z to 1 dot -> length = 3*cols

	// Create OpenGL 3.3 context
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

	// Create window
	GLFWwindow *window = glfwCreateWindow(1800, 600, "Perlin Noise Generator", nullptr, nullptr);
	if(window == nullptr) {
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);

	// Setting up viewport
	glfwSetFramebufferSizeCallback(window, framebuffer_size_callback); // Устанавливаем callback на изменение размеров окна

	// Initialize GLAD
	if(!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
		std::cout << "Failed to initialize GLAD" << std::endl;
		return -1;
	}

	// Calculate Perlin in parallel.
	hipError_t cudaStatus = Perlin1DWithCuda<float>(noise, k, step, numSteps, controlPoints, resultDotsCols, octaveNum);
	if(cudaStatus != hipSuccess) {
		std::cout << stderr << "Perlin1DWithCuda failed!\r\n";
		return -5;
	}

	// Save dots into 3d coords
	for(int i = 0; i < resultDotsCols; i++) {
		vertices[3 * i] = 2 * static_cast<float>(i) / static_cast<float>(resultDotsCols) - 1; // x = 2x(norm)-1, нормализуем и смещаем влево
		vertices[3 * i + 1] = noise[i]; // y
		/*std::cout << "x[" << i << "] = " << vertices[3 * i] << "\t"
					<< "y[" << i << "] = " << vertices[3 * i + 1]	<< "\t"
					<< "z[" << i << "] = " << vertices[3 * i + 2]	<< "\r\n";/**/
	}

	// Create vertex array object.
	uint32_t VAO;
	glGenVertexArrays(1, &VAO);
	std::cout << "Vertex array object have been created with ID = " << VAO << "\r\n";

	// Связываем объект вершинного массива.
	glBindVertexArray(VAO);
	
	// Create vertex buffer object.
	uint32_t VBO;
	glGenBuffers(1, &VBO);
	std::cout << "Vertex buffer object have been created with ID = " << VBO << "\r\n";
	
	// Связываем буфер. Теперь все вызовы буфера с параметром GL_ARRAY_BUFFER
	// будут использоваться для конфигурирования созданного буфера VBO
	glBindBuffer(GL_ARRAY_BUFFER, VBO);

	// Копируем данные вершин в память связанного буфера
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

	// Сообщаем, как OpenGL должен интерпретировать данные вершин,
	// которые мы храним в vertices[]
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void *)0);
	glEnableVertexAttribArray(0);

	// Create vertex shader
	uint32_t vertexShader = glCreateShader(GL_VERTEX_SHADER);

	// Compile vertex shader source code
	glShaderSource(vertexShader, 1, &vertexShaderSource, nullptr);
	glCompileShader(vertexShader);

	// Check vertex shader compile errors
	int32_t success;
	char infoLog[512];
	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
	if(!success) {
		glGetShaderInfoLog(vertexShader, 512, nullptr, infoLog);
		std::cout << "ERROR::SHADER::VERTEX::COMPILATION_FAILED\n" << infoLog << std::endl;
		return -2;
	} else std::cout << "Vertex shader have been compiled!\r\n";

	// Create and compile fragment shader
	uint32_t fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShader, 1, &fragmentShaderSource, nullptr);
	glCompileShader(fragmentShader);

	// Check fragment shader compile errors
	glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &success);
	if(!success) {
		glGetShaderInfoLog(fragmentShader, 512, nullptr, infoLog);
		std::cout << "ERROR::SHADER::FRAGMENT::COMPILATION_FAILED\n" << infoLog << std::endl;
		return -3;
	}
	else std::cout << "Fragment shader have been compiled!\r\n";

	// Создаём объект шейдерной программы
	uint32_t shaderProgram = glCreateProgram();

	// Прикрепляем наши шейдеры к шейдерной программе
	glAttachShader(shaderProgram, vertexShader);
	glAttachShader(shaderProgram, fragmentShader);
	glLinkProgram(shaderProgram);

	// Check shader program linking errors
	glGetProgramiv(shaderProgram, GL_LINK_STATUS, &success);
	if(!success) {
		glGetProgramInfoLog(shaderProgram, 512, nullptr, infoLog);
		std::cout << "ERROR::SHADER::PROGRAM::LINKING_FAILED\n" << infoLog << std::endl;
		return -4;
	}
	else std::cout << "Shader program have been linked!\r\n";

	// Delete the shaders as they're linked into our program now and no longer necessery
	glDeleteShader(vertexShader);
	glDeleteShader(fragmentShader);

	// Create render cycle
	while(!glfwWindowShouldClose(window)) {
		// Input processing
		processInput(window);

		// Rendering
		// Активируем созданный объект
		glUseProgram(shaderProgram);

		// Отменяем связывание???
		glBindVertexArray(VAO);

		// Рисуем ось OX


		// Рисуем шум Перлина
		glDrawArrays(GL_LINE_STRIP, 0, resultDotsCols);

		// Swap buffers
		glfwSwapBuffers(window);
		glfwPollEvents();
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipDeviceReset();
	// glfwTerminate must be called before exiting in order for clean up
	glfwTerminate();
	return 0;
}

/**
* Вспомогательная функция для вычисления шума Перлина на оси с использованием GPU.
*
* \param res – массив с результатом вычисления шума перлина на оси.
* \param k – массив со значениями наклона уравнений в контрольных узлах.
* \param step – величина шага между точками, в которых вычисляется шум.
* \param numSteps – количество точек между контрольными узлами.
* \param controlPoints – количество узлов.
* \param resultDotsCols - количество точек для просчёта.
* 
* \return res – функция изменяет переданный массив.
* \return hipError_t
*/
template<typename T>
hipError_t Perlin1DWithCuda(T *res, const T *k, T step, int numSteps, int controlPoints, int resultDotsCols, int octaveNum) {
	T *dev_res = 0; // pointer to result array in VRAM
	T *dev_octave = 0; // pointer to temp array in VRAM
	T *dev_k = 0; // pointer to array with tilt angle (tg slope angle) in VRAM
	hipError_t cudaStatus;

	// Choose which GPU to run on.
	cudaStatus = hipSetDevice(0);
	if(cudaStatus != hipSuccess) {
		std::cout << stderr << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\r\n";
		goto Error;
	}

	// Allocate GPU buffers for arrays.
	cudaStatus = hipMalloc((void **)&dev_res, resultDotsCols * sizeof(T));
	if(cudaStatus != hipSuccess) {
		std::cout << stderr << "hipMalloc failed!\r\n";
		goto Error;
	}

	// Массив для октав займёт максимально в 2 раза меньше памяти.
	cudaStatus = hipMalloc((void **)&dev_octave, resultDotsCols * sizeof(T) / 2);
	if(cudaStatus != hipSuccess) {
		std::cout << stderr << "hipMalloc failed!\r\n";
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_k, controlPoints * sizeof(T));
	if(cudaStatus != hipSuccess) {
		std::cout << stderr << "hipMalloc failed!\r\n";
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_k, k, controlPoints * sizeof(T), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess) {
		std::cout << stderr << "hipMemcpy failed!\r\n";
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	Perlin1D_kernel<T> <<<1, resultDotsCols>>> (dev_res, dev_octave, dev_k, resultDotsCols, step, resultDotsCols/(controlPoints-1), octaveNum);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess) {
		std::cout << stderr << "addKernel launch failed: %s\n" << hipGetErrorString(cudaStatus) << "\r\n";
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess) {
		std::cout << stderr << "hipDeviceSynchronize returned error code %d after launching addKernel!\n" << cudaStatus << "\r\n";
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(res, dev_res, resultDotsCols * sizeof(T), hipMemcpyDeviceToHost);
	if(cudaStatus != hipSuccess) {
		std::cout << stderr << "hipMemcpy failed!\r\n";
		goto Error;
	}

Error:
	hipFree(dev_res);
	hipFree(dev_k);

	return cudaStatus;
}

// Обработка ресайза окна
void framebuffer_size_callback(GLFWwindow *window, int width, int height) {
	glViewport(0, 0, width, height);
}

// Обработка всех событий ввода: запрос GLFW о нажатии/отпускании клавиш на клавиатуре в данном кадре и соответствующая обработка данных событий
void processInput(GLFWwindow *window) {
	if(glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
		glfwSetWindowShouldClose(window, true);
}